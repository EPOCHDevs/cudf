/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// #define PRINT_DEBUG

#include "reader_impl.hpp"
#include "reader_impl_chunking.hpp"
#include "reader_impl_helpers.hpp"

#include <io/comp/gpuinflate.hpp>
#include <io/comp/nvcomp_adapter.hpp>
#include <io/utilities/config_utils.hpp>

#include <cudf/detail/timezone.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include <algorithm>
#include <iterator>

namespace cudf::io::orc::detail {

namespace {

/**
 * @brief Function that populates column descriptors stream/chunk
 */
std::size_t gather_stream_info_and_update_chunks(
  std::size_t stripe_index,
  std::size_t level,
  orc::StripeInformation const* stripeinfo,
  orc::StripeFooter const* stripefooter,
  host_span<int const> orc2gdf,
  host_span<orc::SchemaType const> types,
  bool use_index,
  bool apply_struct_map,
  std::size_t* num_dictionary_entries,
  std::vector<orc_stream_info>& stream_info,
  cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks)
{
  uint64_t src_offset = 0;
  uint64_t dst_offset = 0;

  auto const get_stream_index_type = [](orc::StreamKind kind) {
    switch (kind) {
      case orc::DATA: return gpu::CI_DATA;
      case orc::LENGTH:
      case orc::SECONDARY: return gpu::CI_DATA2;
      case orc::DICTIONARY_DATA: return gpu::CI_DICTIONARY;
      case orc::PRESENT: return gpu::CI_PRESENT;
      case orc::ROW_INDEX: return gpu::CI_INDEX;
      default:
        // Skip this stream as it's not strictly required
        return gpu::CI_NUM_STREAMS;
    }
  };

  for (auto const& stream : stripefooter->streams) {
    if (!stream.column_id || *stream.column_id >= orc2gdf.size()) {
      dst_offset += stream.length;
      continue;
    }

    auto const column_id = *stream.column_id;
    auto col             = orc2gdf[column_id];

    if (col == -1 and apply_struct_map) {
      // A struct-type column has no data itself, but rather child columns
      // for each of its fields. There is only a PRESENT stream, which
      // needs to be included for the reader.
      auto const schema_type = types[column_id];
      if (not schema_type.subtypes.empty()) {
        if (schema_type.kind == orc::STRUCT && stream.kind == orc::PRESENT) {
          for (auto const& idx : schema_type.subtypes) {
            auto child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
            if (child_idx >= 0) {
              col                             = child_idx;
              auto& chunk                     = chunks[stripe_index][col];
              chunk.strm_id[gpu::CI_PRESENT]  = stream_info.size();
              chunk.strm_len[gpu::CI_PRESENT] = stream.length;
            }
          }
        }
      }
    }
    if (col != -1) {
      if (src_offset >= stripeinfo->indexLength || use_index) {
        auto& chunk           = chunks[stripe_index][col];
        auto const index_type = get_stream_index_type(stream.kind);
        if (index_type < gpu::CI_NUM_STREAMS) {
          chunk.strm_id[index_type]  = stream_info.size();
          chunk.strm_len[index_type] = stream.length;
          // NOTE: skip_count field is temporarily used to track the presence of index streams
          chunk.skip_count |= 1 << index_type;

          if (index_type == gpu::CI_DICTIONARY) {
            chunk.dictionary_start = *num_dictionary_entries;
            chunk.dict_len         = stripefooter->columns[column_id].dictionarySize;
            *num_dictionary_entries += stripefooter->columns[column_id].dictionarySize;
          }
        }
      }

      stream_info.emplace_back(stripeinfo->offset + src_offset,
                               dst_offset,
                               stream.length,
                               stripe_index,
                               level,
                               column_id,
                               stream.kind);
      dst_offset += stream.length;
    }
    src_offset += stream.length;
  }

  return dst_offset;
}

}  // namespace

void reader::impl::query_stripe_compression_info()
{
  if (_file_itm_data->compinfo_ready) { return; }
  if (_selected_columns.num_levels() == 0) { return; }

  auto const rows_to_skip      = _file_itm_data->rows_to_skip;
  auto const rows_to_read      = _file_itm_data->rows_to_read;
  auto const& selected_stripes = _file_itm_data->selected_stripes;

  // If no rows or stripes to read, return empty columns
  // TODO : remove?
  if (rows_to_read == 0 || selected_stripes.empty()) { return; }

  auto& lvl_stripe_data      = _file_itm_data->lvl_stripe_data;
  auto& lvl_data_chunks      = _file_itm_data->lvl_data_chunks;
  auto& lvl_num_dict_entries = _file_itm_data->lvl_num_dict_entries;
  auto& lvl_stripe_is_empty  = _file_itm_data->lvl_stripe_is_empty;

  lvl_stripe_data.resize(_selected_columns.num_levels());
  lvl_data_chunks.resize(_selected_columns.num_levels());
  lvl_num_dict_entries.resize(_selected_columns.num_levels());
  lvl_stripe_is_empty.resize(_selected_columns.num_levels());

  // TODO: Don't have to keep it for all stripe/level. Can reset it after each iter.
  std::unordered_map<stream_id_info, gpu::CompressedStreamInfo*, stream_id_hash, stream_id_equal>
    stream_compinfo_map;

  // Logically view streams as columns
  _file_itm_data->lvl_stream_info.resize(_selected_columns.num_levels());

  // Iterates through levels of nested columns, child column will be one level down
  // compared to parent column.
  auto& col_meta = *_col_meta;
  for (std::size_t level = 0; level < _selected_columns.num_levels(); ++level) {
    auto& columns_level = _selected_columns.levels[level];
    // Association between each ORC column and its cudf::column
    col_meta.orc_col_map.emplace_back(_metadata.get_num_cols(), -1);

    size_type col_id{0};
    for (auto& col : columns_level) {
      // Map each ORC column to its column
      col_meta.orc_col_map[level][col.id] = col_id++;
    }
  }

  for (std::size_t level = 0; level < _selected_columns.num_levels(); ++level) {
    // Get the total number of stripes across all input files.
    std::size_t total_num_stripes =
      std::accumulate(selected_stripes.begin(),
                      selected_stripes.end(),
                      0,
                      [](std::size_t sum, auto& stripe_source_mapping) {
                        return sum + stripe_source_mapping.stripe_info.size();
                      });
    auto& columns_level    = _selected_columns.levels[level];
    auto const num_columns = columns_level.size();
    _file_itm_data->lvl_data_chunks[level] =
      cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>(total_num_stripes, num_columns, _stream);
    auto& chunks = _file_itm_data->lvl_data_chunks[level];
    memset(chunks.base_host_ptr(), 0, chunks.size_bytes());

    auto& stream_info = _file_itm_data->lvl_stream_info[level];
    stream_info.reserve(selected_stripes.size() * selected_stripes.front().stripe_info.size() *
                        num_columns);

    const bool use_index =
      _use_index &&
      // Do stripes have row group index
      _metadata.is_row_grp_idx_present() &&
      // Only use if we don't have much work with complete columns & stripes
      // TODO: Consider nrows, gpu, and tune the threshold
      (rows_to_read > _metadata.get_row_index_stride() && !(_metadata.get_row_index_stride() & 7) &&
       _metadata.get_row_index_stride() > 0 && num_columns * total_num_stripes < 8 * 128) &&
      // Only use if first row is aligned to a stripe boundary
      // TODO: Fix logic to handle unaligned rows
      (rows_to_skip == 0);

    // Tracker for eventually deallocating compressed and uncompressed data
    auto& stripe_data = lvl_stripe_data[level];

    lvl_stripe_is_empty[level].reserve(selected_stripes.size() *
                                       selected_stripes.front().stripe_info.size());

    std::size_t num_dict_entries = 0;
    std::size_t stripe_idx       = 0;

    std::vector<std::pair<std::future<std::size_t>, std::size_t>> read_tasks;
    for (auto const& stripe_source_mapping : selected_stripes) {
      // Iterate through the source files selected stripes
      for (auto const& stripe : stripe_source_mapping.stripe_info) {
        auto const stripe_info   = stripe.first;
        auto const stripe_footer = stripe.second;

        auto stream_count = stream_info.size();
        auto const total_data_size =
          gather_stream_info_and_update_chunks(stripe_idx,
                                               level,
                                               stripe_info,
                                               stripe_footer,
                                               col_meta.orc_col_map[level],
                                               _metadata.get_types(),
                                               use_index,
                                               level == 0,
                                               &num_dict_entries,
                                               stream_info,
                                               chunks);

        lvl_stripe_is_empty[level].push_back(total_data_size == 0);

        //        auto const total_data_size = gather_stream_info(stripe_idx,
        //                                                        level,
        //                                                        stripe_info,
        //                                                        stripe_footer,
        //                                                        col_meta.orc_col_map[level],
        //                                                        _metadata.get_types(),
        //                                                        level == 0,
        //                                                        stream_info);

        auto const is_stripe_data_empty = total_data_size == 0;
        CUDF_EXPECTS(not is_stripe_data_empty or stripe_info->indexLength == 0,
                     "Invalid index rowgroup stream data");

        // Buffer needs to be padded.
        // Required by `copy_uncompressed_kernel`.
        stripe_data.emplace_back(
          cudf::util::round_up_safe(total_data_size, BUFFER_PADDING_MULTIPLE), _stream);
        auto dst_base = static_cast<uint8_t*>(stripe_data.back().data());

        // Coalesce consecutive streams into one read
        while (not is_stripe_data_empty and stream_count < stream_info.size()) {
          auto const d_dst  = dst_base + stream_info[stream_count].dst_pos;
          auto const offset = stream_info[stream_count].offset;
          auto len          = stream_info[stream_count].length;
          stream_count++;

          while (stream_count < stream_info.size() &&
                 stream_info[stream_count].offset == offset + len) {
            len += stream_info[stream_count].length;
            stream_count++;
          }
          if (_metadata.per_file_metadata[stripe_source_mapping.source_idx]
                .source->is_device_read_preferred(len)) {
            read_tasks.push_back(
              std::pair(_metadata.per_file_metadata[stripe_source_mapping.source_idx]
                          .source->device_read_async(offset, len, d_dst, _stream),
                        len));

          } else {
            auto const buffer =
              _metadata.per_file_metadata[stripe_source_mapping.source_idx].source->host_read(
                offset, len);
            CUDF_EXPECTS(buffer->size() == len, "Unexpected discrepancy in bytes read.");
            CUDF_CUDA_TRY(
              hipMemcpyAsync(d_dst, buffer->data(), len, hipMemcpyDefault, _stream.value()));
            _stream.synchronize();
          }
        }

        stripe_idx++;
      }
    }

    for (auto& task : read_tasks) {
      CUDF_EXPECTS(task.first.get() == task.second, "Unexpected discrepancy in bytes read.");
    }

    if (stripe_data.empty()) { continue; }

    lvl_num_dict_entries[level] = num_dict_entries;

    // Setup row group descriptors if using indexes
    if (_metadata.per_file_metadata[0].ps.compression != orc::NONE) {
      auto const& decompressor = *_metadata.per_file_metadata[0].decompressor;
      cudf::detail::hostdevice_vector<gpu::CompressedStreamInfo> compinfo(
        0, stream_info.size(), _stream);

      for (auto const& info : stream_info) {
        compinfo.push_back(gpu::CompressedStreamInfo(
          static_cast<uint8_t const*>(stripe_data[info.stripe_idx].data()) + info.dst_pos,
          info.length));
        stream_compinfo_map[stream_id_info{
          info.stripe_idx, info.level, info.orc_col_idx, info.kind}] =
          &compinfo[compinfo.size() - 1];
#ifdef PRINT_DEBUG
        printf("collec stream [%d, %d, %d, %d]: dst = %lu,  length = %lu\n",
               (int)info.stripe_idx,
               (int)info.level,
               (int)info.orc_col_idx,
               (int)info.kind,
               info.dst_pos,
               info.length);
        fflush(stdout);
#endif
      }

      compinfo.host_to_device_async(_stream);

      gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                     compinfo.size(),
                                     decompressor.GetBlockSize(),
                                     decompressor.GetLog2MaxCompressionRatio(),
                                     _stream);
      compinfo.device_to_host_sync(_stream);

      auto& compinfo_map = _file_itm_data->compinfo_map;
      for (auto& [stream_id, stream_compinfo] : stream_compinfo_map) {
        compinfo_map[stream_id] = {stream_compinfo->num_compressed_blocks,
                                   stream_compinfo->num_uncompressed_blocks,
                                   stream_compinfo->max_uncompressed_size};
#ifdef PRINT_DEBUG
        printf("cache info [%d, %d, %d, %d]:  %lu | %lu | %lu\n",
               (int)stream_id.stripe_idx,
               (int)stream_id.level,
               (int)stream_id.orc_col_idx,
               (int)stream_id.kind,
               (size_t)stream_compinfo->num_compressed_blocks,
               (size_t)stream_compinfo->num_uncompressed_blocks,
               stream_compinfo->max_uncompressed_size);
        fflush(stdout);
#endif
      }

      // Must clear so we will not overwrite the old compression info stream_id.
      stream_compinfo_map.clear();

    } else {
      // printf("no compression \n");
      // fflush(stdout);

      // Set decompressed data size equal to the input size.
      // TODO
    }

    // printf("  end level %d\n\n", (int)level);

  }  // end loop level

  // lvl_stripe_data.clear();
  _file_itm_data->compinfo_ready = true;
}

}  // namespace cudf::io::orc::detail
