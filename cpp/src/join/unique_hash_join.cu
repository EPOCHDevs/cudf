#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "join_common_utils.cuh"
#include "join_common_utils.hpp"

#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/unique_hash_join.cuh>
#include <cudf/hashing/detail/helper_functions.cuh>
#include <cudf/join.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>

namespace cudf {
namespace detail {
namespace {

template <cudf::has_nested HasNested>
auto prepare_device_equal(
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> build,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> probe,
  bool has_nulls,
  cudf::null_equality compare_nulls)
{
  auto const two_table_equal =
    cudf::experimental::row::equality::two_table_comparator(build, probe);
  return comparator_adapter{two_table_equal.equal_to<HasNested == cudf::has_nested::YES>(
    nullate::DYNAMIC{has_nulls}, compare_nulls)};
}

/**
 * @brief Device functor to create a pair of {hash_value, row_index} for a given row.
 *
 * @tparam Hasher The type of internal hasher to compute row hash.
 */
template <typename Hasher>
class build_keys_fn {
 public:
  CUDF_HOST_DEVICE build_keys_fn(Hasher const& hash) : _hash{hash} {}

  __device__ __forceinline__ auto operator()(size_type i) const noexcept
  {
    return cuco::pair{_hash(i), lhs_index_type{i}};
  }

 private:
  Hasher _hash;
};
}  // namespace

template <typename Hasher, cudf::has_nested HasNested>
unique_hash_join<Hasher, HasNested>::unique_hash_join(cudf::table_view const& build,
                                                      cudf::table_view const& probe,
                                                      bool has_nulls,
                                                      cudf::null_equality compare_nulls,
                                                      rmm::cuda_stream_view stream)
  : _has_nulls{has_nulls},
    _is_empty{build.num_rows() == 0},
    _nulls_equal{compare_nulls},
    _build{build},
    _probe{probe},
    _preprocessed_build{
      cudf::experimental::row::equality::preprocessed_table::create(_build, stream)},
    _preprocessed_probe{
      cudf::experimental::row::equality::preprocessed_table::create(_probe, stream)},
    _hash_table{::compute_hash_table_size(build.num_rows()),
                cuco::empty_key{cuco::pair{std::numeric_limits<hash_value_type>::max(),
                                           lhs_index_type{JoinNoneValue}}},
                prepare_device_equal<HasNested>(
                  _preprocessed_build, _preprocessed_probe, has_nulls, compare_nulls),
                {},
                cudf::detail::cuco_allocator{stream},
                stream.value()}
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(0 != this->_build.num_columns(), "Hash join build table is empty");

  if (this->_is_empty) { return; }

  auto const row_hasher = experimental::row::hash::row_hasher{this->_preprocessed_build};
  auto const d_hasher   = row_hasher.device_hasher(nullate::DYNAMIC{this->_has_nulls});

  auto const iter = cudf::detail::make_counting_transform_iterator(0, build_keys_fn{d_hasher});

  size_type const build_table_num_rows{build.num_rows()};
  if (this->_nulls_equal == cudf::null_equality::EQUAL or (not cudf::nullable(this->_build))) {
    this->_hash_table.insert_async(iter, iter + build_table_num_rows, stream.value());
  } else {
    auto stencil = thrust::counting_iterator<size_type>{0};
    auto const row_bitmask =
      cudf::detail::bitmask_and(this->_build, stream, rmm::mr::get_current_device_resource()).first;
    auto const pred = cudf::detail::row_is_valid{row_bitmask};

    // insert valid rows
    this->_hash_table.insert_if_async(
      iter, iter + build_table_num_rows, stencil, pred, stream.value());
  }
}

/*
template <typename Equal, typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
unique_hash_join<Equal, Hasher>::inner_join(std::optional<std::size_t> output_size,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();

  size_type const probe_table_num_rows{this->_probe.num_rows()};

  std::size_t const join_size = output_size ? *output_size : probe_table_num_rows;

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const probe_row_hasher =
    cudf::experimental::row::hash::row_hasher{this->_preprocessed_probe};
  auto const d_probe_hasher = probe_row_hasher.device_hasher(nullate::DYNAMIC{this->_has_nulls});
  return;
}

template <typename Equal, typename Hasher>
std::size_t unique_hash_join<Equal, Hasher>::inner_join_size(cudf::table_view const& probe,
                                                             rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Return directly if build table is empty
  if (_is_empty) { return 0; }

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.");

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);

  return 10;
  cudf::detail::compute_join_output_size(_build,
                                         probe,
                                         _preprocessed_build,
                                         preprocessed_probe,
                                         _hash_table,
                                         cudf::detail::join_kind::INNER_JOIN,
                                         _has_nulls,
                                         _nulls_equal,
                                         stream);
}
*/
}  // namespace detail

template <cudf::has_nested HasNested>
unique_hash_join<HasNested>::~unique_hash_join() = default;

template <cudf::has_nested HasNested>
unique_hash_join<HasNested>::unique_hash_join(cudf::table_view const& build,
                                              cudf::table_view const& probe,
                                              null_equality compare_nulls,
                                              rmm::cuda_stream_view stream)
  : unique_hash_join(build, probe, nullable_join::YES, compare_nulls, stream)
{
}

template <cudf::has_nested HasNested>
unique_hash_join<HasNested>::unique_hash_join(cudf::table_view const& build,
                                              cudf::table_view const& probe,
                                              nullable_join has_nulls,
                                              null_equality compare_nulls,
                                              rmm::cuda_stream_view stream)
  : _impl{std::make_unique<impl_type>(
      build, probe, has_nulls == nullable_join::YES, compare_nulls, stream)}
{
}

template <cudf::has_nested HasNested>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
unique_hash_join<HasNested>::inner_join(std::optional<std::size_t> output_size,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr) const
{
  return _impl->inner_join(output_size, stream, mr);
}

template <cudf::has_nested HasNested>
std::size_t unique_hash_join<HasNested>::inner_join_size(rmm::cuda_stream_view stream) const
{
  return _impl->inner_join_size(stream);
}

}  // namespace cudf
